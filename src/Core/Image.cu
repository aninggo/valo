#include "hip/hip_runtime.h"
// Copyright © 2016 Mikko Ronkainen <firstname@mikkoronkainen.com>
// License: MIT, see the LICENSE file.

#include <vector>

#include "tinyformat/tinyformat.h"

#include "stb/stb_image.h"
#include "stb/stb_image_write.h"

#include "Core/Common.h"
#include "Core/Image.h"
#include "App.h"
#include "Utils/Allocator.h"
#include "Utils/Log.h"
#include "Utils/StringUtils.h"
#include "Math/MathUtils.h"
#include "Filters/Filter.h"

using namespace Raycer;

Image::Image()
{
}

Image::~Image()
{
	RAYCER_FREE(pixels);
}

Image::Image(uint32_t length_)
{
	resize(length_);
}

Image::Image(uint32_t width_, uint32_t height_)
{
	resize(width_, height_);
}

Image::Image(uint32_t width_, uint32_t height_, float* rgbaData)
{
	load(width_, height_, rgbaData);
}

Image::Image(const std::string& fileName)
{
	load(fileName);
}

void Image::load(uint32_t width_, uint32_t height_, float* rgbaData)
{
	resize(width_, height_);

	for (uint32_t i = 0; i < length; ++i)
	{
		uint32_t dataIndex = i * 4;

		pixels[i].r = rgbaData[dataIndex];
		pixels[i].g = rgbaData[dataIndex + 1];
		pixels[i].b = rgbaData[dataIndex + 2];
		pixels[i].a = rgbaData[dataIndex + 3];
	}
}

void Image::load(const std::string& fileName)
{
	App::getLog().logInfo("Loading image from %s", fileName);

	if (stbi_is_hdr(fileName.c_str()))
	{
		int32_t newWidth, newHeight, components;
		float* loadData = stbi_loadf(fileName.c_str(), &newWidth, &newHeight, &components, 3); // RGB

		if (loadData == nullptr)
			throw std::runtime_error(tfm::format("Could not load HDR image file: %s", stbi_failure_reason()));

		resize(uint32_t(newWidth), uint32_t(newHeight));

		for (uint32_t y = 0; y < height; ++y)
		{
			for (uint32_t x = 0; x < width; ++x)
			{
				uint32_t pixelIndex = y * width + x;
				uint32_t dataIndex = (height - 1 - y) * width * 3 + x * 3; // flip vertically

				pixels[pixelIndex].r = loadData[dataIndex];
				pixels[pixelIndex].g = loadData[dataIndex + 1];
				pixels[pixelIndex].b = loadData[dataIndex + 2];
				pixels[pixelIndex].a = 1.0f;
			}
		}

		stbi_image_free(loadData);
	}
	else
	{
		int32_t newWidth, newHeight, components;
		uint32_t* loadData = reinterpret_cast<uint32_t*>(stbi_load(fileName.c_str(), &newWidth, &newHeight, &components, 4)); // RGBA

		if (loadData == nullptr)
			throw std::runtime_error(tfm::format("Could not load image file: %s", stbi_failure_reason()));

		resize(uint32_t(newWidth), uint32_t(newHeight));

		for (uint32_t y = 0; y < height; ++y)
		{
			for (uint32_t x = 0; x < width; ++x)
				pixels[y * width + x] = Color::fromAbgrValue(loadData[(height - 1 - y) * width + x]); // flip vertically
		}

		stbi_image_free(loadData);
	}
}

void Image::save(const std::string& fileName, bool writeToLog) const
{
	if (writeToLog)
		App::getLog().logInfo("Saving image to %s", fileName);

	int32_t result = 0;

	if (StringUtils::endsWith(fileName, ".png") || StringUtils::endsWith(fileName, ".bmp") || StringUtils::endsWith(fileName, ".tga"))
	{
		std::vector<uint32_t> saveData(length);

		for (uint32_t y = 0; y < height; ++y)
		{
			for (uint32_t x = 0; x < width; ++x)
				saveData[(height - 1 - y) * width + x] = pixels[y * width + x].clamped().getAbgrValue(); // flip vertically
		}

		if (StringUtils::endsWith(fileName, ".png"))
			result = stbi_write_png(fileName.c_str(), int32_t(width), int32_t(height), 4, &saveData[0], int32_t(width * sizeof(uint32_t)));
		else if (StringUtils::endsWith(fileName, ".bmp"))
			result = stbi_write_bmp(fileName.c_str(), int32_t(width), int32_t(height), 4, &saveData[0]);
		else if (StringUtils::endsWith(fileName, ".tga"))
			result = stbi_write_tga(fileName.c_str(), int32_t(width), int32_t(height), 4, &saveData[0]);
	}
	else if (StringUtils::endsWith(fileName, ".hdr"))
	{
		std::vector<float> saveData(length * 3);

		for (uint32_t y = 0; y < height; ++y)
		{
			for (uint32_t x = 0; x < width; ++x)
			{
				uint32_t dataIndex = (height - 1 - y) * width * 3 + x * 3; // flip vertically
				uint32_t pixelIndex = y * width + x;

				saveData[dataIndex] = float(pixels[pixelIndex].r);
				saveData[dataIndex + 1] = float(pixels[pixelIndex].g);
				saveData[dataIndex + 2] = float(pixels[pixelIndex].b);
			}
		}

		result = stbi_write_hdr(fileName.c_str(), int32_t(width), int32_t(height), 3, &saveData[0]);
	}
	else
		throw std::runtime_error("Could not save the image (non-supported format)");

	if (result == 0)
		throw std::runtime_error(tfm::format("Could not save the image: %s", stbi_failure_reason()));
}

void Image::resize(uint32_t length_)
{
	resize(length_, 1);
}

void Image::resize(uint32_t width_, uint32_t height_)
{
	width = width_;
	height = height_;
	length = width * height;

	RAYCER_FREE(pixels);
	pixels = static_cast<Color*>(RAYCER_MALLOC(length * sizeof(Color)));

	if (pixels == nullptr)
		throw std::runtime_error("Could not allocate memory for image");

	clear();
}

void Image::setPixel(uint32_t x, uint32_t y, const Color& color)
{
	pixels[y * width + x] = color;
}

void Image::setPixel(uint32_t index, const Color& color)
{
	pixels[index] = color;
}

void Image::clear()
{
	memset(pixels, 0, length * sizeof(Color));
}

void Image::clear(const Color& color)
{
	for (uint32_t i = 0; i < length; ++i)
		pixels[i] = color;
}

void Image::applyGamma(float gamma)
{
	for (uint32_t i = 0; i < length; ++i)
		pixels[i] = Color::pow(pixels[i], gamma).clamped();
}

void Image::applyFastGamma(float gamma)
{
	for (uint32_t i = 0; i < length; ++i)
		pixels[i] = Color::fastPow(pixels[i], gamma).clamped();
}

void Image::swapComponents()
{
	for (uint32_t i = 0; i < length; ++i)
	{
		Color c2 = pixels[i];

		pixels[i].r = c2.a;
		pixels[i].g = c2.b;
		pixels[i].b = c2.g;
		pixels[i].a = c2.r;
	}
}

void Image::fillWithTestPattern()
{
	for (uint32_t y = 0; y < height; ++y)
	{
		for (uint32_t x = 0; x < width; ++x)
		{
			Color color = Color::black();

			if (x % 2 == 0 && y % 2 == 0)
				color = Color::lerp(Color::red(), Color::blue(), float(x) / float(width));

			pixels[y * width + x] = color;
		}
	}
}

CUDA_CALLABLE uint32_t Image::getWidth() const
{
	return width;
}

CUDA_CALLABLE uint32_t Image::getHeight() const
{
	return height;
}

CUDA_CALLABLE uint32_t Image::getLength() const
{
	return length;
}

CUDA_CALLABLE Color Image::getPixel(uint32_t x, uint32_t y) const
{
	assert(x < width && y < height);

	return pixels[y * width + x];
}

CUDA_CALLABLE Color Image::getPixel(uint32_t index) const
{
	assert(index < length);

	return pixels[index];
}

CUDA_CALLABLE Color Image::getPixelNearest(float u, float v) const
{
	uint32_t x = uint32_t(u * float(width - 1) + 0.5f);
	uint32_t y = uint32_t(v * float(height - 1) + 0.5f);

	return getPixel(x, y);
}

CUDA_CALLABLE Color Image::getPixelBilinear(float u, float v) const
{
	float x = u * float(width - 1);
	float y = v * float(height - 1);

	uint32_t ix = uint32_t(x);
	uint32_t iy = uint32_t(y);

	float tx2 = x - float(ix);
	float ty2 = y - float(iy);

	tx2 = MathUtils::smoothstep(tx2);
	ty2 = MathUtils::smoothstep(ty2);

	float tx1 = 1.0f - tx2;
	float ty1 = 1.0f - ty2;

	uint32_t ix1 = ix + 1;
	uint32_t iy1 = iy + 1;

	if (ix1 > width - 1)
		ix1 = width - 1;

	if (iy1 > height - 1)
		iy1 = height - 1;

	Color c11 = getPixel(ix, iy);
	Color c21 = getPixel(ix1, iy);
	Color c12 = getPixel(ix, iy1);
	Color c22 = getPixel(ix1, iy1);

	// bilinear interpolation
	return (tx1 * c11 + tx2 * c21) * ty1 + (tx1 * c12 + tx2 * c22) * ty2;
}

CUDA_CALLABLE Color Image::getPixelBicubic(float u, float v, Filter& filter) const
{
	float x = u * float(width - 1);
	float y = v * float(height - 1);

	int32_t ix = int32_t(x);
	int32_t iy = int32_t(y);

	float fx = x - float(ix);
	float fy = y - float(iy);

	Color cumulativeColor;
	float cumulativeFilterWeight = 0.0f;

	for (int32_t oy = -1; oy <= 2; oy++)
	{
		for (int32_t ox = -1; ox <= 2; ox++)
		{
			int32_t sx = ix + ox;
			int32_t sy = iy + oy;

			if (sx < 0)
				sx = 0;

			if (sx > int32_t(width - 1))
				sx = int32_t(width - 1);

			if (sy < 0)
				sy = 0;

			if (sy > int32_t(height - 1))
				sy = int32_t(height - 1);

			Color color = getPixel(uint32_t(sx), uint32_t(sy));
			float filterWeight = filter.getWeight(Vector2(float(ox) - fx, float(oy) - fy));

			cumulativeColor += color * filterWeight;
			cumulativeFilterWeight += filterWeight;
		}
	}

	return cumulativeColor / cumulativeFilterWeight;
}

Color* Image::getPixelData()
{
	return pixels;
}

const Color* Image::getPixelData() const
{
	return pixels;
}
