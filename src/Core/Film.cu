#include "hip/hip_runtime.h"
﻿// Copyright © 2016 Mikko Ronkainen <firstname@mikkoronkainen.com>
// License: MIT, see the LICENSE file.

#include "Precompiled.h"

#include <GL/gl3w.h>

#ifdef USE_CUDA
#include <cuda_gl_interop.h>
#include <>
#endif

#include "Core/Common.h"
#include "App.h"
#include "Core/Film.h"
#include "Tonemappers/Tonemapper.h"
#include "Utils/Log.h"
#include "Utils/CudaUtils.h"
#include "Utils/GLUtils.h"

using namespace Raycer;

Film::Film(bool windowed_) : windowed(windowed_)
{
}

void Film::initialize()
{
	if (windowed)
	{
		glGenTextures(1, &textureId);

		GLUtils::checkError("Could not create OpenGL texture");

		glBindTexture(GL_TEXTURE_2D, textureId);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

		GLUtils::checkError("Could not set OpenGL texture parameters");
	}
}

void Film::shutdown()
{
	if (windowed)
	{
#ifdef USE_CUDA

		if (textureResource != nullptr)
		{
			CudaUtils::checkError(hipGraphicsUnregisterResource(textureResource), "Could not unregister OpenGL texture");
			textureResource = nullptr;
		}

#endif

		glDeleteTextures(1, &textureId);

		GLUtils::checkError("Could not delete OpenGL texture");
	}
}

void Film::resize(uint32_t width_, uint32_t height_)
{
	width = width_;
	height = height_;
	length = width * height;

	App::getLog().logInfo("Resizing film to %sx%s", width, height);

	cumulativeImage.resize(width, height);
	normalizedImage.resize(width, height);
	tonemappedImage.resize(width, height);

	if (windowed)
	{
#ifdef USE_CUDA

		if (textureResource != nullptr)
		{
			CudaUtils::checkError(hipGraphicsUnregisterResource(textureResource), "Could not unregister OpenGL texture");
			textureResource = nullptr;
		}

#endif

		glBindTexture(GL_TEXTURE_2D, textureId);
		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, GLsizei(width), GLsizei(height), 0, GL_RGBA, GL_FLOAT, nullptr);
		glBindTexture(GL_TEXTURE_2D, 0);

		GLUtils::checkError("Could not reserve OpenGL texture memory");

#ifdef USE_CUDA
		CudaUtils::checkError(hipGraphicsGLRegisterImage(&textureResource, textureId, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore), "Could not register OpenGL texture");
#endif
	}
}

void Film::clear(RendererType type)
{
	cumulativeImage.clear(type);
	pixelSamples = 0;
	cleared = true;
}

bool Film::hasBeenCleared() const
{
	return cleared;
}

void Film::resetCleared()
{
	cleared = false;
}

CUDA_CALLABLE void Film::addSample(uint32_t x, uint32_t y, const Color& color, float filterWeight)
{
	Color temp = cumulativeImage.getPixel(x, y);

	temp.r += color.r * filterWeight;
	temp.g += color.g * filterWeight;
	temp.b += color.b * filterWeight;
	temp.a += filterWeight;

	cumulativeImage.setPixel(x, y, temp);
}

CUDA_CALLABLE void Film::addSample(uint32_t index, const Color& color, float filterWeight)
{
	Color temp = cumulativeImage.getPixel(index);

	temp.r += color.r * filterWeight;
	temp.g += color.g * filterWeight;
	temp.b += color.b * filterWeight;
	temp.a += filterWeight;

	cumulativeImage.setPixel(index, temp);
}

#ifdef USE_CUDA

__global__ void normalizeKernel(hipSurfaceObject_t cumulative, hipSurfaceObject_t normalized, uint32_t width, uint32_t height)
{
	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= width || y >= height)
		return;

	float4 color;
	surf2Dread(&color, cumulative, x * sizeof(float4), y);

	color.x /= color.w;
	color.y /= color.w;
	color.z /= color.w;
	color.w = 1.0f;

	surf2Dwrite(color, normalized, x * sizeof(float4), y);
}

#endif

void Film::normalize(RendererType type)
{
	if (type == RendererType::CPU)
	{
		#pragma omp parallel for
		for (int32_t i = 0; i < int32_t(length); ++i)
		{
			Color color = cumulativeImage.getPixel(i);
			color /= color.a;
			color.a = 1.0f;

			normalizedImage.setPixel(i, color);
		}
	}
	else
	{
#ifdef USE_CUDA

		dim3 dimBlock(16, 16);
		dim3 dimGrid;

		dimGrid.x = (width + dimBlock.x - 1) / dimBlock.x;
		dimGrid.y = (height + dimBlock.y - 1) / dimBlock.y;

		normalizeKernel<<<dimGrid, dimBlock>>>(cumulativeImage.getSurfaceObject(), normalizedImage.getSurfaceObject(), width, height);
		CudaUtils::checkError(hipPeekAtLastError(), "Could not launch normalize kernel");
		CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute normalize kernel");

#endif
	}
}

#ifdef USE_CUDA

__global__ void tonemapKernel(hipSurfaceObject_t input, hipSurfaceObject_t output, uint32_t width, uint32_t height)
{
	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= width || y >= height)
		return;

	float4 color;
	surf2Dread(&color, input, x * sizeof(float4), y);

	color.x = pow(color.x, 1.0f / 2.2f);
	color.y = pow(color.y, 1.0f / 2.2f);
	color.z = pow(color.z, 1.0f / 2.2f);
	color.w = 1.0f;

	surf2Dwrite(color, output, x * sizeof(float4), y);
}

#endif

void Film::tonemap(Tonemapper& tonemapper, RendererType type)
{
	if (type == RendererType::CPU)
	{
		tonemapper.apply(normalizedImage, tonemappedImage);
	}
	else
	{
#ifdef USE_CUDA

		dim3 dimBlock(16, 16);
		dim3 dimGrid;

		dimGrid.x = (width + dimBlock.x - 1) / dimBlock.x;
		dimGrid.y = (height + dimBlock.y - 1) / dimBlock.y;

		tonemapKernel<<<dimGrid, dimBlock>>>(normalizedImage.getSurfaceObject(), tonemappedImage.getSurfaceObject(), width, height);
		CudaUtils::checkError(hipPeekAtLastError(), "Could not launch tonemap kernel");
		CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute tonemap kernel");

#endif
	}
}

#ifdef USE_CUDA

__global__ void updateTextureKernel(hipSurfaceObject_t input, hipSurfaceObject_t output, uint32_t width, uint32_t height)
{
	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= width || y >= height)
		return;

	float4 color;
	surf2Dread(&color, input, x * sizeof(float4), y);
	surf2Dwrite(color, output, x * sizeof(float4), y);
}

#endif

void Film::updateTexture(RendererType type)
{
	if (windowed)
	{
		if (type == RendererType::CPU)
		{
			glBindTexture(GL_TEXTURE_2D, textureId);
			glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, GLsizei(width), GLsizei(height), GL_RGBA, GL_FLOAT, tonemappedImage.getData());
			glBindTexture(GL_TEXTURE_2D, 0);

			GLUtils::checkError("Could not upload OpenGL texture data");
		}
		else
		{
#ifdef USE_CUDA

			CudaUtils::checkError(hipGraphicsMapResources(1, &textureResource, 0), "Could not map texture resource");

			hipArray_t textureData;
			CudaUtils::checkError(hipGraphicsSubResourceGetMappedArray(&textureData, textureResource, 0, 0), "Could not get mapped array");

			hipResourceDesc resDesc;
			memset(&resDesc, 0, sizeof(resDesc));
			resDesc.resType = hipResourceTypeArray;
			resDesc.res.array.array = textureData;

			hipSurfaceObject_t surfaceObject;
			CudaUtils::checkError(hipCreateSurfaceObject(&surfaceObject, &resDesc), "Could not create surface object");
			
			dim3 dimBlock(16, 16);
			dim3 dimGrid;

			dimGrid.x = (width + dimBlock.x - 1) / dimBlock.x;
			dimGrid.y = (height + dimBlock.y - 1) / dimBlock.y;

			updateTextureKernel<<<dimGrid, dimBlock>>>(tonemappedImage.getSurfaceObject(), surfaceObject, width, height);
			CudaUtils::checkError(hipPeekAtLastError(), "Could not launch update texture kernel");
			CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute update texture kernel");

			CudaUtils::checkError(hipDestroySurfaceObject(surfaceObject), "Could not destroy surface object");
			CudaUtils::checkError(hipGraphicsUnmapResources(1, &textureResource, 0), "Could not unmap texture resource");

#endif
		}
	}
}

Color Film::getCumulativeColor(uint32_t x, uint32_t y) const
{
	return cumulativeImage.getPixel(x, y);
}

Color Film::getNormalizedColor(uint32_t x, uint32_t y) const
{
	return normalizedImage.getPixel(x, y);
}

Color Film::getTonemappedColor(uint32_t x, uint32_t y) const
{
	return tonemappedImage.getPixel(x, y);
}

CUDA_CALLABLE Image& Film::getCumulativeImage()
{
	return cumulativeImage;
}

CUDA_CALLABLE Image& Film::getNormalizedImage()
{
	return normalizedImage;
}

CUDA_CALLABLE Image& Film::getTonemappedImage()
{
	return tonemappedImage;
}

CUDA_CALLABLE uint32_t Film::getWidth() const
{
	return width;
}

CUDA_CALLABLE uint32_t Film::getHeight() const
{
	return height;
}

CUDA_CALLABLE uint32_t Film::getLength() const
{
	return length;
}

GLuint Film::getTextureId() const
{
	return textureId;
}
