// Copyright © 2016 Mikko Ronkainen <firstname@mikkoronkainen.com>
// License: MIT, see the LICENSE file.

#include "Precompiled.h"

#ifdef USE_CUDA
#include <hip/hip_runtime.h>
#endif

#include "Core/Common.h"
#include "Core/CudaAlloc.h"
#include "Utils/CudaUtils.h"
#include "Core/Scene.h"
#include "Core/Film.h"
#include "Math/Random.h"

using namespace Raycer;

template <typename T>
CudaAlloc<T>::CudaAlloc(bool pinned_) : pinned(pinned_)
{
}

template <typename T>
CudaAlloc<T>::~CudaAlloc()
{
	release();
}

template <typename T>
void CudaAlloc<T>::resize(size_t count)
{
	assert(count > 0);

	release();

	maxCount = count;

#ifdef USE_CUDA

	if (pinned)
	{
		CudaUtils::checkError(hipHostMalloc(&hostPtr, sizeof(T) * count), "Could not allocate pinned host memory");

		if (hostPtr == nullptr)
			throw std::runtime_error("Could not allocate pinned host memory");
	}
	else
	{
		hostPtr = static_cast<T*>(malloc(sizeof(T) * count));

		if (hostPtr == nullptr)
			throw std::runtime_error("Could not allocate host memory");
	}

	CudaUtils::checkError(hipMalloc(&devicePtr, sizeof(T) * count), "Could not allocate device memory");

	if (devicePtr == nullptr)
		throw std::runtime_error("Could not allocate device memory");

#else

	hostPtr = static_cast<T*>(malloc(sizeof(T) * count));

	if (hostPtr == nullptr)
		throw std::runtime_error("Could not allocate host memory");

#endif
}

template <typename T>
void CudaAlloc<T>::write(T* source, size_t count)
{
	assert(count <= maxCount);

	memcpy(hostPtr, source, sizeof(T) * count);

#ifdef USE_CUDA
	CudaUtils::checkError(hipMemcpy(devicePtr, hostPtr, sizeof(T) * count, hipMemcpyHostToDevice), "Could not write data to device");
#endif
}

template <typename T>
void CudaAlloc<T>::read(size_t count)
{
	(void)count;
	assert(count <= maxCount);

#ifdef USE_CUDA
	CudaUtils::checkError(hipMemcpy(hostPtr, devicePtr, sizeof(T) * count, hipMemcpyDeviceToHost), "Could not read data from device");
#endif
}

template <typename T>
CUDA_CALLABLE T* CudaAlloc<T>::getPtr() const
{
#ifdef USE_CUDA
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
	return devicePtr;
#else
	return hostPtr;
#endif
#else
	return hostPtr;
#endif
}

template <typename T>
T* CudaAlloc<T>::getHostPtr() const
{
	return hostPtr;
}

template <typename T>
T* CudaAlloc<T>::getDevicePtr() const
{
	return devicePtr;
}

template <typename T>
void CudaAlloc<T>::release()
{
	maxCount = 0;

#ifdef USE_CUDA

	if (hostPtr != nullptr)
	{
		if (pinned)
			CudaUtils::checkError(hipHostFree(hostPtr), "Could not free pinned host memory");
		else
			free(hostPtr);

		hostPtr = nullptr;
	}

	if (devicePtr != nullptr)
	{
		CudaUtils::checkError(hipFree(devicePtr), "Could not free device memory");
		devicePtr = nullptr;
	}

#else

	if (hostPtr != nullptr)
	{
		free(hostPtr);
		hostPtr = nullptr;
	}

#endif
}

template class CudaAlloc<uint32_t>;
template class CudaAlloc<Scene>;
template class CudaAlloc<Film>;
template class CudaAlloc<Image>;
template class CudaAlloc<Texture>;
template class CudaAlloc<Material>;
template class CudaAlloc<Triangle>;
template class CudaAlloc<BVHNode>;
template class CudaAlloc<BVHNodeSOA<4>>;
template class CudaAlloc<BVHNodeSOA<8>>;
template class CudaAlloc<BVHNodeSOA<16>>;
template class CudaAlloc<TriangleSOA<4>>;
template class CudaAlloc<TriangleSOA<8>>;
template class CudaAlloc<TriangleSOA<16>>;
template class CudaAlloc<RandomGeneratorState>;
