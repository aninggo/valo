#include "hip/hip_runtime.h"
// Copyright © 2016 Mikko Ronkainen <firstname@mikkoronkainen.com>
// License: MIT, see the LICENSE file.

#include "Precompiled.h"

#include "Textures/MarbleTexture.h"
#include "Math/Vector2.h"
#include "Math/Vector3.h"
#include "Math/Color.h"

using namespace Raycer;

void MarbleTexture::initialize()
{
	noise.initialize(seed);
}

CUDA_CALLABLE Color MarbleTexture::getColor(const Vector2& texcoord, const Vector3& position) const
{
	(void)texcoord;

	float n1 = std::abs(std::cos(position.x * density + noise.getFbmNoise(8, 2.0f, 0.5f, position.x * 2.0f, position.y * 2.0f, position.z * 2.0f) * swirliness));
	n1 = (1.0f - n1) / transparency;

	Color streakColor1(streakColor);
	streakColor1.a = n1;

	return Color::alphaBlend(marbleColor, streakColor1);
}
