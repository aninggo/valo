#include "hip/hip_runtime.h"
// Copyright © 2016 Mikko Ronkainen <firstname@mikkoronkainen.com>
// License: MIT, see the LICENSE file.

#include "Precompiled.h"

#include "Filters/BellFilter.h"
#include "Math/Vector2.h"

using namespace Raycer;

namespace
{
	CUDA_CALLABLE float calculateWeight(float s)
	{
		s = std::abs(s);

		if (s < 0.5f)
			return 0.75f - (s * s);
		
		if (s <= 1.5f)
			return 0.5f * std::pow(s - 1.5f, 2.0f);
		
		return 0.0f;
	}
}

CUDA_CALLABLE float BellFilter::getWeight(float s) const
{
	return calculateWeight(s);
}

CUDA_CALLABLE float BellFilter::getWeight(const Vector2& point) const
{
	return calculateWeight(point.x) * calculateWeight(point.y);
}

CUDA_CALLABLE Vector2 BellFilter::getRadius() const
{
	return Vector2(1.5f, 1.5f);
}
