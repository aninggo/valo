#include "hip/hip_runtime.h"
// Copyright © 2016 Mikko Ronkainen <firstname@mikkoronkainen.com>
// License: MIT, see the LICENSE file.

#include "Precompiled.h"

#include "Filters/BellFilter.h"
#include "Math/Vector2.h"

using namespace Valo;

namespace
{
	CUDA_CALLABLE float calculateWeight(float s)
	{
		s = std::abs(s);

		if (s < 0.5f)
			return 0.75f - (s * s);
		
		if (s <= 1.5f)
		{
			s = s - 1.5f;
			return 0.5f * (s * s);
		}
		
		return 0.0f;
	}
}

CUDA_CALLABLE float BellFilter::getWeight(float s) const
{
	return calculateWeight(s);
}

CUDA_CALLABLE float BellFilter::getWeight(const Vector2& point) const
{
	return calculateWeight(point.x) * calculateWeight(point.y);
}

CUDA_CALLABLE Vector2 BellFilter::getRadius() const
{
	return Vector2(1.5f, 1.5f);
}
