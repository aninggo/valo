#include "hip/hip_runtime.h"
// Copyright © 2016 Mikko Ronkainen <firstname@mikkoronkainen.com>
// License: MIT, see the LICENSE file.

#include "Precompiled.h"

#include "Core/Intersection.h"
#include "Core/Ray.h"
#include "Core/Scene.h"
#include "Integrators/Integrator.h"
#include "Materials/Material.h"
#include "Math/Random.h"

using namespace Raycer;

CUDA_CALLABLE Color Integrator::calculateLight(const Scene& scene, const Intersection& intersection, const Ray& ray, Random& random) const
{
	switch (type)
	{
		case IntegratorType::PATH: return pathIntegrator.calculateLight(scene, intersection, ray, random);
		case IntegratorType::DOT: return dotIntegrator.calculateLight(scene, intersection, ray, random);
		case IntegratorType::AMBIENT_OCCLUSION: return aoIntegrator.calculateLight(scene, intersection, ray, random);
		case IntegratorType::DIRECT_LIGHT: return directIntegrator.calculateLight(scene, intersection, ray, random);
		default: return Color::black();
	}
}

std::string Integrator::getName() const
{
	switch (type)
	{
		case IntegratorType::PATH: return "path";
		case IntegratorType::DOT: return "dot";
		case IntegratorType::AMBIENT_OCCLUSION: return "ao";
		case IntegratorType::DIRECT_LIGHT: return "direct";
		default: return "unknown";
	}
}

CUDA_CALLABLE bool Integrator::getRandomEmissiveIntersection(const Scene& scene, const Intersection& origin, Random& random, Intersection& emissiveIntersection)
{
	if (scene.getEmissiveTrianglesCount() == 0)
		return false;

	const Triangle& triangle = scene.getEmissiveTriangles()[random.getUint32(0, scene.getEmissiveTrianglesCount() - 1)];
	Intersection triangleIntersection = triangle.getRandomIntersection(scene, random);
	Vector3 originToTriangle = triangleIntersection.position - origin.position;
	float distance2 = originToTriangle.lengthSquared();
	float distance = std::sqrt(distance2);
	Vector3 direction = originToTriangle / distance;

	Ray visibilityRay;
	visibilityRay.origin = origin.position;
	visibilityRay.direction = direction;
	visibilityRay.minDistance = scene.general.rayMinDistance;
	visibilityRay.maxDistance = distance - scene.general.rayMinDistance;
	visibilityRay.isVisibilityRay = true;
	visibilityRay.precalculate();

	Intersection visibilityIntersection;
	
	if (!scene.intersect(visibilityRay, visibilityIntersection))
	{
		emissiveIntersection = triangleIntersection;
		return true;
	}

	return false;
}

CUDA_CALLABLE DirectLightSample Integrator::calculateDirectLightSample(const Scene& scene, const Intersection& origin, const Intersection& emissiveIntersection)
{
	Vector3 originToEmissive = emissiveIntersection.position - origin.position;
	float distance2 = originToEmissive.lengthSquared();
	float distance = std::sqrt(distance2);
	Vector3 direction = originToEmissive / distance;

	float cosine = direction.dot(-emissiveIntersection.normal);

	if (cosine < 0.0f)
		return DirectLightSample();

	const Material& emissiveMaterial = scene.getMaterial(emissiveIntersection.materialIndex);

	DirectLightSample directLightSample;
	directLightSample.emittance = emissiveMaterial.getEmittance(scene, emissiveIntersection.texcoord, emissiveIntersection.position);
	directLightSample.direction = direction;
	directLightSample.pdf = (1.0f / scene.getEmissiveTrianglesCount()) * (1.0f / emissiveIntersection.area) * (distance2 / cosine);
	directLightSample.visible = true;

	return directLightSample;
}

float Integrator::calculateBalanceHeuristic(uint32_t nf, float fPdf, uint32_t ng, float gPdf)
{
	return (nf * fPdf) / (nf * fPdf + ng * gPdf);
}

float Integrator::calculatePowerHeuristic(uint32_t nf, float fPdf, uint32_t ng, float gPdf)
{
	float f = nf * fPdf;
	float g = ng * gPdf;

	return (f * f) / (f * f + g * g);
}
