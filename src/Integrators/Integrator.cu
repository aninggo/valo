#include "hip/hip_runtime.h"
﻿// Copyright © 2016 Mikko Ronkainen <firstname@mikkoronkainen.com>
// License: MIT, see the LICENSE file.

#include "Precompiled.h"

#include "Core/Intersection.h"
#include "Core/Ray.h"
#include "Core/Scene.h"
#include "Integrators/Integrator.h"
#include "Materials/Material.h"
#include "Math/Random.h"

using namespace Raycer;

CUDA_CALLABLE Color Integrator::calculateLight(const Scene& scene, const Intersection& intersection, const Ray& ray, Random& random) const
{
	switch (type)
	{
		case IntegratorType::PATH: return pathIntegrator.calculateLight(scene, intersection, ray, random);
		case IntegratorType::DOT: return dotIntegrator.calculateLight(scene, intersection, ray, random);
		case IntegratorType::AMBIENT_OCCLUSION: return aoIntegrator.calculateLight(scene, intersection, ray, random);
		case IntegratorType::DIRECT_LIGHT: return directIntegrator.calculateLight(scene, intersection, ray, random);
		default: return Color::black();
	}
}

std::string Integrator::getName() const
{
	switch (type)
	{
		case IntegratorType::PATH: return "path";
		case IntegratorType::DOT: return "dot";
		case IntegratorType::AMBIENT_OCCLUSION: return "ao";
		case IntegratorType::DIRECT_LIGHT: return "direct";
		default: return "unknown";
	}
}

Intersection Integrator::getRandomEmissiveIntersection(const Scene& scene, Random& random)
{
	const Triangle& triangle = scene.getEmissiveTriangles()[random.getUint32(0, scene.getEmissiveTrianglesCount() - 1)];
	return triangle.getRandomIntersection(scene, random);
}

bool Integrator::isIntersectionVisible(const Scene& scene, const Intersection& origin, const Intersection& emissiveIntersection)
{
	Vector3 originToEmissive = emissiveIntersection.position - origin.position;
	float distance = originToEmissive.length();
	Vector3 direction = originToEmissive / distance;

	Ray visibilityRay;
	visibilityRay.origin = origin.position;
	visibilityRay.direction = direction;
	visibilityRay.minDistance = scene.general.rayMinDistance;
	visibilityRay.maxDistance = distance - scene.general.rayMinDistance;
	visibilityRay.isVisibilityRay = true;
	visibilityRay.precalculate();

	Intersection visibilityIntersection;
	return !scene.intersect(visibilityRay, visibilityIntersection);
}

CUDA_CALLABLE DirectLightSample Integrator::calculateDirectLightSample(const Scene& scene, const Intersection& origin, const Intersection& emissiveIntersection)
{
	Vector3 originToEmissive = emissiveIntersection.position - origin.position;
	float distance2 = originToEmissive.lengthSquared();
	float distance = std::sqrt(distance2);

	DirectLightSample result;
	result.direction = originToEmissive / distance;
	result.originCosine = result.direction.dot(origin.normal);
	result.lightCosine = result.direction.dot(-emissiveIntersection.normal);

	if (result.originCosine <= 0.0f || result.lightCosine <= 0.0f)
	{
		result.visible = false;
		return result;
	}

	const Material& emissiveMaterial = scene.getMaterial(emissiveIntersection.materialIndex);

	result.emittance = emissiveMaterial.getEmittance(scene, emissiveIntersection.texcoord, emissiveIntersection.position);
	result.lightPdf = (1.0f / scene.getEmissiveTrianglesCount()) * (1.0f / emissiveIntersection.area) * (distance2 / result.lightCosine);
	result.visible = true;

	return result;
}

float Integrator::balanceHeuristic(uint32_t nf, float fPdf, uint32_t ng, float gPdf)
{
	return (nf * fPdf) / (nf * fPdf + ng * gPdf);
}

float Integrator::powerHeuristic(uint32_t nf, float fPdf, uint32_t ng, float gPdf)
{
	float f = nf * fPdf;
	float g = ng * gPdf;

	return (f * f) / (f * f + g * g);
}
