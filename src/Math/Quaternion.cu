#include "hip/hip_runtime.h"
// Copyright © 2016 Mikko Ronkainen <firstname@mikkoronkainen.com>
// License: MIT, see the LICENSE file.

#include <cassert>

#include "Math/Quaternion.h"
#include "Math/AxisAngle.h"
#include "Math/Matrix4x4.h"
#include "Math/Vector3.h"
#include "Math/MathUtils.h"

using namespace Raycer;

CUDA_CALLABLE Quaternion::Quaternion(float w_, float x_, float y_, float z_) : w(w_), x(x_), y(y_), z(z_)
{
}

CUDA_CALLABLE Quaternion::Quaternion(const AxisAngle& axisAngle) : Quaternion(axisAngle.axis, axisAngle.angle)
{
}

CUDA_CALLABLE Quaternion::Quaternion(const Vector3& axis, float angle)
{
	assert(axis.isNormal());

	float radiansPer2 = MathUtils::degToRad(angle) / 2.0f;
	float cosine = std::cos(radiansPer2);
	float sine = std::sin(radiansPer2);

	w = cosine;
	x = axis.x * sine;
	y = axis.y * sine;
	z = axis.z * sine;
}

namespace Raycer
{
	CUDA_CALLABLE Quaternion operator+(const Quaternion& q1, const Quaternion& q2)
	{
		return Quaternion(q1.w + q2.w, q1.x + q2.x, q1.y + q2.y, q1.z + q2.z);
	}

	CUDA_CALLABLE Quaternion operator-(const Quaternion& q1, const Quaternion& q2)
	{
		return Quaternion(q1.w - q2.w, q1.x - q2.x, q1.y - q2.y, q1.z - q2.z);
	}

	CUDA_CALLABLE Quaternion operator*(const Quaternion& q1, const Quaternion& q2)
	{
		Quaternion r;

		r.w = q1.w * q2.w - q1.x * q2.x - q1.y * q2.y - q1.z * q2.z;
		r.x = q1.w * q2.x + q1.x * q2.w + q1.y * q2.z - q1.z * q2.y;
		r.y = q1.w * q2.y - q1.x * q2.z + q1.y * q2.w + q1.z * q2.x;
		r.z = q1.w * q2.z + q1.x * q2.y - q1.y * q2.x + q1.z * q2.w;

		return r;
	}

	CUDA_CALLABLE Quaternion operator*(const Quaternion& q, float s)
	{
		return Quaternion(q.w * s, q.x * s, q.y * s, q.z * s);
	}

	CUDA_CALLABLE Quaternion operator*(float s, const Quaternion& q)
	{
		return q * s;
	}

	CUDA_CALLABLE Quaternion operator/(const Quaternion& q, float s)
	{
		float invS = 1.0f / s;
		return Quaternion(q.w * invS, q.x * invS, q.y * invS, q.z * invS);
	}

	CUDA_CALLABLE Quaternion operator-(const Quaternion& q)
	{
		return Quaternion(-q.w, -q.x, -q.y, -q.z);
	}

	CUDA_CALLABLE bool operator==(const Quaternion& q1, const Quaternion& q2)
	{
		return MathUtils::almostSame(q1.w, q2.w) && MathUtils::almostSame(q1.x, q2.x) && MathUtils::almostSame(q1.y, q2.y) && MathUtils::almostSame(q1.z, q2.z);
	}

	CUDA_CALLABLE bool operator!=(const Quaternion& q1, const Quaternion& q2)
	{
		return !(q1 == q2);
	}
}

CUDA_CALLABLE Quaternion& Quaternion::operator+=(const Quaternion& q)
{
	*this = *this + q;
	return *this;
}

CUDA_CALLABLE Quaternion& Quaternion::operator-=(const Quaternion& q)
{
	*this = *this - q;
	return *this;
}

CUDA_CALLABLE Quaternion& Quaternion::operator*=(const Quaternion& q)
{
	*this = *this * q;
	return *this;
}

CUDA_CALLABLE Quaternion& Quaternion::operator*=(float s)
{
	*this = *this * s;
	return *this;
}

CUDA_CALLABLE Quaternion& Quaternion::operator/=(float s)
{
	*this = *this / s;
	return *this;
}

CUDA_CALLABLE Vector3 Quaternion::rotate(const Vector3& v) const
{
	Vector3 r(x, y, z);
	return v + 2.0f * r.cross(r.cross(v) + w * v);
}

CUDA_CALLABLE float Quaternion::length() const
{
	return std::sqrt(w * w + x * x + y * y + z * z);
}

CUDA_CALLABLE float Quaternion::lengthSquared() const
{
	return (w * w + x * x + y * y + z * z);
}

CUDA_CALLABLE void Quaternion::conjugate()
{
	x = -x;
	y = -y;
	z = -z;
}

CUDA_CALLABLE Quaternion Quaternion::conjugated() const
{
	return Quaternion(w, -x, -y, -z);
}

CUDA_CALLABLE void Quaternion::normalize()
{
	*this /= length();
}

CUDA_CALLABLE Quaternion Quaternion::normalized() const
{
	return *this / length();
}

CUDA_CALLABLE bool Quaternion::isZero() const
{
	return (w == 0.0f && x == 0.0f && y == 0.0f && z == 0.0f);
}

bool Quaternion::isNan() const
{
	return (std::isnan(w) || std::isnan(x) || std::isnan(y) || std::isnan(z));
}

CUDA_CALLABLE bool Quaternion::isNormal() const
{
	return MathUtils::almostSame(lengthSquared(), 1.0f);
}

CUDA_CALLABLE float Quaternion::dot(const Quaternion& q) const
{
	return (w * q.w) + (x * q.x) + (y * q.y) + (z * q.z);
}

CUDA_CALLABLE AxisAngle Quaternion::toAxisAngle() const
{
	assert(isNormal());

	AxisAngle aa;

	aa.angle = MathUtils::radToDeg(2.0f * acos(w));
	float sine2 = 1.0f - w * w;

	if (sine2 < FLT_EPSILON)
	{
		aa.axis.x = 1.0f;
		aa.axis.y = 0.0f;
		aa.axis.z = 0.0f;
	}
	else
	{
		float invSine = 1.0f / sqrt(sine2);

		aa.axis.x = x * invSine;
		aa.axis.y = y * invSine;
		aa.axis.z = z * invSine;
	}

	return aa;
}

CUDA_CALLABLE Matrix4x4 Quaternion::toMatrix4x4() const
{
	assert(isNormal());

	Matrix4x4 result(
		1.0f - 2.0f * y * y - 2.0f * z * z, 2.0f * x * y - 2.0f * w * z, 2.0f * x * z + 2.0f * w * y, 0.0f,
		2.0f * x * y + 2.0f * w * z, 1.0f - 2.0f * x * x - 2.0f * z * z, 2.0f * y * z + 2.0f * w * x, 0.0f,
		2.0f * x * z - 2.0f * w * y, 2.0f * y * z - 2.0f * w * x, 1.0f - 2.0f * x * x - 2.0f * y * y, 0.0f,
		0.0f, 0.0f, 0.0f, 1.0f);

	return result;
}

CUDA_CALLABLE Quaternion Quaternion::lerp(const Quaternion& q1, const Quaternion& q2, float t)
{
	assert(t >= 0.0f && t <= 1.0f);
	return q1 * (1.0f - t) + q2 * t;
}

CUDA_CALLABLE Quaternion Quaternion::slerp(const Quaternion& q1, const Quaternion& q2, float t)
{
	assert(t >= 0.0f && t <= 1.0f);

	float cosine = q1.dot(q2) / (q1.length() * q2.length());
	float theta = acos(cosine);
	float sinTheta = sin(theta);

	if (sinTheta < FLT_EPSILON)
		return lerp(q1, q2, t);
	
	Quaternion tq1 = (cosine < 0.0f) ? -q1 : q1;
	Quaternion q = sin((1.0f - t) * theta) * tq1 + sin(t * theta) * q2;

	return q / sinTheta;
}
